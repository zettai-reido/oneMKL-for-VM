#include "hip/hip_runtime.h"
#include "device_error_handler.hpp"
#include "scalar_args.hpp"
#include "strider.hpp"
#include "vector_args.hpp"
#include "xtypes.hpp"

#include "scalar.hpp"

namespace vecma::detail {


template <class Functor, bool is_cyclic, bool has_error_handler>
__global__
void kernel(VectorArgs va, Strider str, Functor const& fop = {}, DeviceErrorHandler const& deh = {}) {
    const size_t index = static_cast<size_t>(threadIdx.x) + blockIdx.x * blockDim.x;
    if (index >= str.get_neval()) { return; }

    using TypeIn = typename Functor::TypeIn;
    using TypeC = typename Functor::TypeC;
    using TypeOut = typename Functor::TypeOut;
    constexpr int n_i = Functor::n_i;
    constexpr int n_c = Functor::n_c;
    constexpr int n_o = Functor::n_o; 

    (void)n_c;

    ScalarArgs sa;
    sa = va.template gather<TypeIn, TypeC, n_i, n_c, is_cyclic>(str, index);

    Status e_status = fop(sa);
    if constexpr (has_error_handler) {
        Status st = fop.math_check(e_status, sa);
        if (st != Status::kSuccess) { 
            deh.template action<TypeOut, n_o>(sa, st);
            sa.st = st;
        }
        va.template scatter<TypeOut, n_o, true>(sa, str, index);
    } else {
        va.template scatter<TypeOut, n_o, false>(sa, str, index);
    } // if constexpr(has_error_handler)
}

template <class Functor>
Status evaluate(VectorArgs& va, Strider str, ErrorHandler const& eh) {
    constexpr int n_i = Functor::n_i;
    constexpr int n_o = Functor::n_o; 

    using TypeOut = typename Functor::TypeOut;


    str.template precompute<n_i, n_o>();

    size_t neval = str.get_neval();
    int bs = 256;
    int nb = (neval + bs - 1) / bs;

    if (str.is_cyclic()) {
        if (eh.is_enabled()) {
            DeviceErrorHandler deh (eh);
            kernel<Functor, true, true> <<<nb, bs>>>(va, str, Functor(), deh); 
        } else {
            DeviceErrorHandler deh;
            kernel<Functor, true, false> <<<nb, bs>>>(va, str, Functor(), deh); 
        }
    } else {
        kernel<Functor, false, false> <<<nb, bs>>>(va, str, Functor());
    }
    return Status::kSuccess;
};

Status evaluate_pow_h(VectorArgs& va, Strider str, ErrorHandler const& eh) { return evaluate<scalar::PowH>(va, str, eh); }
Status evaluate_pow_s(VectorArgs& va, Strider str, ErrorHandler const& eh) { return evaluate<scalar::PowS>(va, str, eh); }
Status evaluate_pow_d(VectorArgs& va, Strider str, ErrorHandler const& eh) { return evaluate<scalar::PowD>(va, str, eh); }
Status evaluate_pow_c(VectorArgs& va, Strider str, ErrorHandler const& eh) { return evaluate<scalar::PowC>(va, str, eh); }
Status evaluate_pow_z(VectorArgs& va, Strider str, ErrorHandler const& eh) { return evaluate<scalar::PowZ>(va, str, eh); }

} // namespace vecma::detail

